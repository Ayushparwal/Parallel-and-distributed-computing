/*
   Program name: 2vecAddUsingManyBlocks(N>threads).cu
    Author name: Dr. Nileshchandra Pikle
           Email: nilesh.pikle@gmail.com
  Contact Number: 7276834418 
         Webpage: https://piklenileshchandra.wixsite.com/personal    

    Purpose: To perform vector addition using CUDA using multiple thread blocks
             Assume Total number of threads launched is less than N 

    Description: Vector addition using multiple thread blocks
    As there might be some situations where number of threads launched more than available number of
    elements in the array. In such scenarios threads may access invalid memory location that is beyond 
    the size of array.
    for example if array size is 4 and you launched 5 threads then 5th thread access invalid memory    
    location as array size is 4. To handle this condition limit thread execution by parametre (gid < 4)
    This condition ensures that only 4 threads will be executing the vector addition code.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>
#include<math.h>


__global__ void vecAdd_kernel_UsingManyBlocks(int *a, int *b, int *c, int N)
{
  
  int gid = threadIdx.x + blockIdx.x * blockDim.x; // Global thread Id
  int stride = gridDim.x * blockDim.x;
   for(int i = gid; i < N; i+= stride )// To avoid out of order memory access when  " # threads > N "
   {
     c[i] = a[i] + b[i];
   }
}
void vecAddCPU(int *a, int *b, int *c, int N)
{
  int i;
  for(i = 0; i < N; i++)
  {
    c[i] = a[i] + b[i];
  }
}

int main()
{
 int i,N = 2<<24; 
  
  /*********************** Memory Allocation on CPU **********************************/
  
  int *h_a = (int *)malloc(N * sizeof(int)); // Memory allocation on CPU for vector h_a  input
  int *h_b = (int *)malloc(N * sizeof(int)); // Memory allocation on CPU for vector h_b  input
  int *h_c = (int *)malloc(N * sizeof(int)); // Memory allocation on CPU for vector h_c  output
  int *hr_c = (int *)malloc(N * sizeof(int));// Memory allocation on CPU for vector hr_c  output from GPU

  int *d_a, *d_b, *d_c; // Decleration of GPU variables
   
  /****************************** Error Handling ***********************************/
   if(!h_a)
   {
    printf("\n CPU: Error occured while allocating memory to h_a!");
   }
   if(!h_b)
   {
    printf("\n CPU: Error occured while allocating memory to h_b!");
   }
   if(!h_c)
   {
    printf("\n CPU: Error occured while allocating memory to h_c!");
   }

  /************************** Data initialization on CPU *****************************/

   for(i = 0; i < N; i++)
   {
     h_a[i] = 2;
     h_b[i] = 2;
     h_c[i] = 0;
     hr_c[i] = 0;
   }   
  /************************** Vector addition on CPU *********************************/
   clock_t begin =  clock();
   
   vecAddCPU(h_a, h_b, h_c,N);
   clock_t end =  clock();
   double CPUtime_taken = (double)(end - begin)/CLOCKS_PER_SEC;
   /*
   for(i = 0; i < N; i++)
   {
     printf("\n h_c[%d] = %d",i,h_c[i]);
   }
   */

  /*
    1. To perform vector addition on GPU first we have to declare variables on GPU.
       These vaiables have been already declared as d_a, d_b, d_c where d_ stands for
       these varaibles are declared on device. Note that d_ is NOT a keyword it used 
       to separate device and host variables.

    2. As program will be executed on GPU, memory should be allocated for device variables as well
       The memory is allocated using cudaMalloc() function which takes two arguments address of device
       variable and size of memory.

    3. Once memory is allocated om GPU, data should be initialized. Either you can initialize data by 
       using separate kernel or transfer it from host to device. In this assignment we will use
       data transfer because it is often used in real life applications.
       
       To transfer data cudaMemcpy() function is used which has 4 parameters as follows
       
       cudaMemcpy(dest_addr, src_addr, size, direction_of_copy);
  
       a. dest_addr = Destination variable address
       b. src_addr  = Source variable address
       c. size      = Size of memory to be transfered
       b. direction_of_copy = data can be transfered from Host to Device of Device to Host

    4. Kernel lauch is similar to C-programming function. Only difference is thread configuraion is added.
 
       vecAdd_kernel<<<NumberOfThreadBlocks, NumberOfThreadsPerBlock>>>(d_a, d_b, d_c, N);

       NumberOfThreadBlocks   : Integer value determines number of thread blocks to be launched
       NumberOfThreadsPerBlock: Integer value determines number of threads per block to be launched

    5. Data is copiedback from device to host using cudaMemcpy() function 
  
    6. Finally both CPU and GPU memories are freed     
  */

 
  /************************** 2. Memory allocation on GPU ******************************/

   hipMalloc((void **)&d_a, N*sizeof(int)); // Allocate memory on GPU for variable d_a  input
   hipMalloc((void **)&d_b, N*sizeof(int)); // Allocate memory on GPU for variable d_b  input
   hipMalloc((void **)&d_c, N*sizeof(int)); // Allocate memory on GPU for variable d_c  output

  /******************** 3. Transfer data from Host to Device ***************************/

  hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice); // Copy data from Host to Device for vector a
  hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice); // Copy data from Host to Device for vector b
  hipMemcpy(d_c, hr_c, N*sizeof(int), hipMemcpyHostToDevice); // Copy data from Host to Device for vector c

  /******************* 4. Kernel lauch to execute vector addition on Device **********************/
  int numThreadsPerBlock, numThreadBlocks;
  numThreadsPerBlock = 128;
  numThreadBlocks = ceil(N/(float)numThreadsPerBlock);
  /* Declaring Time variables to measure GPU time*/
  float GPUelapsed = 0.0;  // To store final kernel execution time 
  hipEvent_t start, stop; // Variables to record start and stop of kernel
  hipEventCreate(&start); // Event create start
  hipEventCreate(&stop);  // Event create stop

  hipEventRecord(start, 0); // Record time at start variables

  /*Vector Addition using many thread blocks*/
 
  vecAdd_kernel_UsingManyBlocks<<<numThreadBlocks, numThreadsPerBlock>>>(d_a, d_b, d_c, N);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&GPUelapsed, start, stop);

  hipEventDestroy(start); // Event destroy start
  hipEventDestroy(stop);  // Event destroy stop
  printf("\n *****************************************************************************\n");
  printf("\n Vector addition using many blocks and #threads <= N");
  printf("\n N = %d   # threads = %d",N,numThreadBlocks*numThreadsPerBlock);
  printf("\n Elapsed Time on CPU is %f ms", CPUtime_taken*1000);
  printf("\n Elapsed Time on GPU is %0.10f ms",GPUelapsed);
  printf("\n Speedup = (CPU time)/(GPU time) = %f", (CPUtime_taken*1000)/GPUelapsed);
  printf("\n *****************************************************************************\n");
 
  /********************* 5. Copy results back from GPU to CPU************************************/
  hipMemcpy(hr_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost); // Copy data from Device to Host for vector c
  
  /************* Validate whether CPU and GPU results are matching or NOT ***********************/
  for(i = 0; i < N; i++)
  {
   //printf("\n hr_c[%d] = %d", i, hr_c[i]);
   if(hr_c[i] != h_c[i])
   {
    printf("\n Results are wrong!!!\n");
   }  

   }
  /* 6. Free CPU memory*/
  free(h_a); 
  free(h_b);
  free(h_c);
  free(hr_c);
  /* 7. Free GPU memory*/
  hipFree(d_a); 
  hipFree(d_b);
  hipFree(d_c);
return(0);
}


